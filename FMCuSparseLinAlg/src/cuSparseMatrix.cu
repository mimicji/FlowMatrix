#include <cuSparseMatrix.cuh>

using namespace FlowMatrix;

cuSparseMatrix::cuSparseMatrix()
{
    this->matDescr = 0;

    this->num_rows = -1;
    this->num_cols = -1;
    this->nnz = -1;

    this->d_csrOffsets = NULL;
    this->d_cooOffsets = NULL;
    this->d_columns = NULL;
    this->d_values = NULL;

    this->h_csrOffsets = NULL;
    this->h_cooOffsets = NULL;
    this->h_columns = NULL;
    this->h_values = NULL;
}

cuSparseMatrix::cuSparseMatrix(int num_rows, int num_cols)
{
    this->matDescr = 0;

    this->num_rows = num_rows;
    this->num_cols = num_cols;
    this->nnz = -1;

    this->d_csrOffsets = NULL;
    this->d_cooOffsets = NULL;
    this->d_columns = NULL;
    this->d_values = NULL;

    this->h_csrOffsets = NULL;
    this->h_cooOffsets = NULL;
    this->h_columns = NULL;
    this->h_values = NULL;
}

cuSparseMatrix::cuSparseMatrix(hipStream_t &stream, int num_rows, int num_cols, int nnz, int *csrOffsets, int *columns, float *values)
{
    int *d_csrOffsets, *d_columns;
    float *d_values;

    CHECK_CUDA( hipMallocAsync((void**) &d_csrOffsets,
                           (num_rows + 1) * sizeof(int), stream) );
    CHECK_CUDA( hipMallocAsync((void**) &d_columns, nnz * sizeof(int), stream)   );
    CHECK_CUDA( hipMallocAsync((void**) &d_values,  nnz * sizeof(float), stream) );

    CHECK_CUDA( hipMemcpyAsync(d_csrOffsets, csrOffsets, (num_rows + 1)  * sizeof(int),
            hipMemcpyHostToDevice, stream) );
    CHECK_CUDA( hipMemcpyAsync(d_columns, columns, nnz * sizeof(int),
            hipMemcpyHostToDevice, stream) );
    CHECK_CUDA( hipMemcpyAsync(d_values, values,
            nnz * sizeof(float), hipMemcpyHostToDevice, stream) );
    this->setMatrixByDevicePtrs(num_rows, num_cols, nnz, d_csrOffsets, d_columns, d_values);
}

void cuSparseMatrix::fromDB(hipsparseHandle_t &handle, int num_rows, int num_cols, int nnz, const int *cooOffsets, const int *columns, const float *values)
{
    this->num_rows = num_rows;
    this->num_cols = num_cols;
    this->nnz = nnz;

    if (nnz <= 0) return;

    hipStream_t stream; 
    CHECK_CUSPARSE(hipsparseGetStream(handle, &stream));

    CHECK_CUDA( hipMallocAsync((void**) &this->d_cooOffsets, 
        (this->nnz) * sizeof(int), stream));
    CHECK_CUDA( hipMemcpyAsync(this->d_cooOffsets, cooOffsets,
        (this->nnz) * sizeof(int),
        hipMemcpyHostToDevice, stream) );

    CHECK_CUDA( hipMallocAsync((void**) &this->d_columns, 
        (this->nnz) * sizeof(int), stream));
    CHECK_CUDA( hipMemcpyAsync(this->d_columns, columns, this->nnz * sizeof(int),
        hipMemcpyHostToDevice, stream) );
    
    CHECK_CUDA( hipMallocAsync((void**) &this->d_values, 
        (this->nnz) * sizeof(float), stream));
    CHECK_CUDA( hipMemcpyAsync(this->d_values, values, this->nnz * sizeof(float),
        hipMemcpyHostToDevice, stream) );    

    this->toCsr(handle);
}

void cuSparseMatrix::setMatrixByDevicePtrs(int num_rows, int num_cols, int nnz, int *d_csrOffsets, int *d_columns, float *d_values)
{
    if (d_csrOffsets == NULL || d_columns == NULL || d_values == NULL)
    {
        printf("Warning: Set matrix desc with NULL array!\n");
    }

    this->num_rows = num_rows;
    this->num_cols = num_cols;
    this->nnz = nnz;

    this->d_csrOffsets = d_csrOffsets;
    this->d_columns = d_columns;
    this->d_values = d_values;
    this->d_cooOffsets = NULL;

    this->h_csrOffsets = NULL;
    this->h_cooOffsets = NULL;
    this->h_columns = NULL;
    this->h_values = NULL;

    CHECK_CUSPARSE(hipsparseCreateCsr(&this->matDescr, num_rows, num_cols, nnz,
        d_csrOffsets, d_columns, d_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
}

void cuSparseMatrix::setMainDiagonal(hipsparseHandle_t &handle, int num_rows, int num_cols, int nnz, int *indices, float value /*=1.0f*/)
{
    this->nnz = nnz;
    this->num_rows = num_rows;
    this->num_cols = num_cols;

    this->h_cooOffsets = new int[nnz];
    this->h_columns = new int[nnz];
    this->h_values = new float[nnz];

    memcpy(this->h_cooOffsets, indices, sizeof(int)*nnz);
    memcpy(this->h_columns, indices, sizeof(int)*nnz);
    std::fill_n(this->h_values, nnz, value); 

    // Transfer to GPU and convert to CSR
    hipStream_t stream; 
    CHECK_CUSPARSE(hipsparseGetStream(handle, &stream));
    this->toDevice(stream);
    this->toCsr(handle);
}

void cuSparseMatrix::toCsr(hipsparseHandle_t &handle, bool keepCoo /*= false*/)
{
    if (UNLIKELY(this->d_csrOffsets != NULL))
    {
        printf("[W] Matrix is already in CSR format.\n");
        return;
    }
    assert(this->d_cooOffsets != NULL);
    hipStream_t stream; 
    CHECK_CUSPARSE(hipsparseGetStream(handle, &stream));
    CHECK_CUDA( hipMallocAsync((void**) &this->d_csrOffsets,
        (this->num_rows + 1) * sizeof(int), stream) );
    CHECK_CUSPARSE(hipsparseXcoo2csr(handle, this->d_cooOffsets, this->nnz,
        this->num_rows, this->d_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO));
    
    // Set Matrix descriptor
    CHECK_CUSPARSE(hipsparseCreateCsr(&this->matDescr, 
        this->num_rows, this->num_cols, this->nnz,
        this->d_csrOffsets, this->d_columns, this->d_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));  
    
    if (!keepCoo)
    {
        CHECK_CUDA( hipFreeAsync(this->d_cooOffsets, stream) );
        this->d_cooOffsets = NULL;
    }
}

void cuSparseMatrix::toCoo(hipsparseHandle_t &handle, bool keepCsr /* = false */)
{
    if (UNLIKELY(this->d_cooOffsets != NULL))
    {
        printf("[W] Matrix is already in COO format.\n");
        return;
    }
    assert(this->d_csrOffsets != NULL);
    hipStream_t stream; 
    CHECK_CUSPARSE(hipsparseGetStream(handle, &stream));
    CHECK_CUDA( hipMallocAsync((void**) &this->d_cooOffsets,
        (this->nnz) * sizeof(int), stream) );
    CHECK_CUSPARSE(hipsparseXcsr2coo(handle, this->d_csrOffsets, this->nnz,
        this->num_rows, this->d_cooOffsets, HIPSPARSE_INDEX_BASE_ZERO));
    if (!keepCsr)
    {
        CHECK_CUDA( hipFreeAsync(this->d_csrOffsets, stream) );
        this->d_csrOffsets = NULL;
    }
}

void cuSparseMatrix::toDevice(hipStream_t &stream)
{
    // First, we transfer Rowptr or RowIdx, depending on the format
    assert(this->h_csrOffsets != NULL || this->h_cooOffsets != NULL);
    if (this->h_csrOffsets != NULL)
    {
        CHECK_CUDA( hipMallocAsync((void**) &this->d_csrOffsets, 
            (this->num_rows + 1) * sizeof(int), stream));
        CHECK_CUDA( hipMemcpyAsync(this->d_csrOffsets, this->h_csrOffsets,
            (this->num_rows + 1) * sizeof(int),
            hipMemcpyHostToDevice, stream) );
        delete [] this->h_csrOffsets;
        this->h_csrOffsets = NULL;
    }
    if (this->h_cooOffsets != NULL)
    {
        CHECK_CUDA( hipMallocAsync((void**) &this->d_cooOffsets, 
            (this->nnz) * sizeof(int), stream));
        CHECK_CUDA( hipMemcpyAsync(this->d_cooOffsets, this->h_cooOffsets,
            (this->nnz) * sizeof(int),
            hipMemcpyHostToDevice, stream) );
        delete [] this->h_cooOffsets;
        this->h_cooOffsets = NULL;
    }

    // Transfer col and values
    assert(this->h_columns != NULL && this->h_values != NULL);

    CHECK_CUDA( hipMallocAsync((void**) &this->d_columns, 
        (this->nnz) * sizeof(int), stream));
    CHECK_CUDA( hipMallocAsync((void**) &this->d_values, 
        (this->nnz) * sizeof(float), stream));

    CHECK_CUDA( hipMemcpyAsync(this->d_columns, this->h_columns, this->nnz * sizeof(int),
        hipMemcpyHostToDevice, stream) );
    CHECK_CUDA( hipMemcpyAsync(this->d_values, this->h_values, this->nnz * sizeof(float),
        hipMemcpyHostToDevice, stream) );    

    // Set Matrix descriptor
    if (this->d_csrOffsets != NULL)
    {
        CHECK_CUSPARSE(hipsparseCreateCsr(&this->matDescr, 
            this->num_rows, this->num_cols, this->nnz,
            this->d_csrOffsets, this->d_columns, this->d_values,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F)); 
    }

    // Reset ptrs
    delete [] this->h_columns;
    delete [] this->h_values;
    this->h_columns = NULL;
    this->h_values = NULL;
}

void cuSparseMatrix::toHost(hipStream_t &stream, bool keepOnDevice /*= false*/)
{
    // First, we transfer Rowptr or RowIdx, depending on the format
    assert(this->d_csrOffsets != NULL || this->d_cooOffsets != NULL);
    if (this->d_csrOffsets != NULL)
    {
        this->h_csrOffsets = new int[this->num_rows + 1];
        CHECK_CUDA( hipMemcpyAsync(this->h_csrOffsets, this->d_csrOffsets,
            (this->num_rows + 1) * sizeof(int),
            hipMemcpyDeviceToHost, stream) );
        if (!keepOnDevice)
        {
            CHECK_CUDA( hipFreeAsync(this->d_csrOffsets, stream) );
            this->d_csrOffsets = NULL;
        }
    }

    if (this->d_cooOffsets != NULL)
    {
        this->h_cooOffsets = new int[this->nnz];
        CHECK_CUDA( hipMemcpyAsync(this->h_cooOffsets, this->d_cooOffsets,
            (this->nnz) * sizeof(int),
            hipMemcpyDeviceToHost, stream) );
        if (!keepOnDevice)
        {
            CHECK_CUDA( hipFreeAsync(this->d_cooOffsets, stream) );
            this->d_cooOffsets = NULL;
        }

    }

    // Transfer col and values
    assert(this->d_columns != NULL && this->d_values != NULL);

    this->h_columns = new int[this->nnz];
    this->h_values = new float[this->nnz];

    CHECK_CUDA( hipMemcpyAsync(this->h_columns, this->d_columns, this->nnz * sizeof(int),
            hipMemcpyDeviceToHost, stream) );

    CHECK_CUDA( hipMemcpyAsync(this->h_values, this->d_values, this->nnz * sizeof(float),
            hipMemcpyDeviceToHost, stream) );

    // Free CUDA memory
    if (!keepOnDevice)
    {
        if (this->matDescr != 0) CHECK_CUSPARSE( hipsparseDestroySpMat(this->matDescr) );
        CHECK_CUDA( hipFreeAsync(this->d_columns, stream) );
        CHECK_CUDA( hipFreeAsync(this->d_values, stream) );
        this->d_columns = NULL;
        this->d_values = NULL;
    }

    // Reset ptrs
    this->matDescr = 0;
}
void cuSparseMatrix::freeDeviceMem(hipStream_t &stream)
{
    if (this->d_csrOffsets != NULL)
    {
        CHECK_CUDA( hipFreeAsync(this->d_csrOffsets, stream) );
        this->d_csrOffsets = NULL;
    }
    if (this->d_cooOffsets != NULL)
    {
        CHECK_CUDA( hipFreeAsync(this->d_cooOffsets, stream) );
        this->d_cooOffsets = NULL;
    }
    if (this->d_columns != NULL)
    {
        CHECK_CUDA( hipFreeAsync(this->d_columns, stream) );
        this->d_columns = NULL;
    }
    if (this->d_values != NULL)
    {
        CHECK_CUDA( hipFreeAsync(this->d_values, stream) );
        this->d_values = NULL;
    }
    if (this->matDescr != 0)
    {
        CHECK_CUSPARSE( hipsparseDestroySpMat(this->matDescr) );
        this->matDescr = 0;
    }
}


cuSparseMatrix::~cuSparseMatrix()
{
    if (this->d_csrOffsets != NULL)
    {
        CHECK_CUDA( hipFree(this->d_csrOffsets) );
        this->d_csrOffsets = NULL;
    }
    if (this->d_cooOffsets != NULL)
    {
        CHECK_CUDA( hipFree(this->d_cooOffsets) );
        this->d_cooOffsets = NULL;
    }
    if (this->d_columns != NULL)
    {
        CHECK_CUDA( hipFree(this->d_columns) );
        this->d_columns = NULL;
    }
    if (this->d_values != NULL)
    {
        CHECK_CUDA( hipFree(this->d_values) );
        this->d_values = NULL;
    }
    if (this->matDescr != 0)
    {
        CHECK_CUSPARSE( hipsparseDestroySpMat(this->matDescr) );
        this->matDescr = 0;
    }
    if (this->h_csrOffsets != NULL)
    {
        delete[] this->h_csrOffsets;
    }
    if (this->h_cooOffsets != NULL)
    {
        delete[] this->h_cooOffsets;
    }
    if (this->h_columns != NULL)
    {
        delete[] this->h_columns;
    }
    if (this->h_values != NULL)
    {
        delete[] this->h_values;
    }
}

void cuSparseMatrix::print() const
{
    std::cout << "Shape: " << this->num_rows << " * " << this->num_cols << std::endl;
    std::cout << "NNZ: " << this->nnz << std::endl;
    std::cout << "Values:"<< std::endl;
    if (this->nnz >0 && (this->h_cooOffsets == NULL || this->h_columns == NULL || this->h_values == NULL))
    {
        std::cout << "  Error! This matrix is not ready for printing." << std::endl;
        return;
    }
    if (nnz >0)
    {
        for (int i=0; i<this->nnz; i++)
        {
            std::cout << "  (" << this->h_cooOffsets[i] << ", " 
                << this->h_columns[i] << ") "
                << this->h_values[i];
            if (i%4 == 3)
            {
                std::cout << std::endl;
            }
            else
            {
                std::cout << "    ";
            }
        }
        if (this->nnz%4 != 0) std::cout << std::endl;
    }
    else
    {
        std::cout << "  (None)" << std::endl;
    }

    return;
}