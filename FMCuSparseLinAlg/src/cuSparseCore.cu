#include <cuSparseCore.cuh>

using namespace FlowMatrix;

CuSparseCore::CuSparseCore(bool always_use_safe)
{
    // Create stream
    CHECK_CUDA(hipStreamCreate(&this->stream));
    CHECK_CUDA(hipStreamCreate(&this->associate_stream));

    // Create handle
    CHECK_CUSPARSE( hipsparseCreate(&this->handle) );
    CHECK_CUSPARSE(hipsparseSetStream(this->handle, this->stream));
    CHECK_CUSPARSE( hipsparseCreate(&this->associate_handle));
    CHECK_CUSPARSE(hipsparseSetStream(this->associate_handle, this->associate_stream));

    // Create buffer
    this->buffer1_size = BUFFER_INIT_SIZE;
    CHECK_CUDA(hipMallocAsync((void**) &this->buffer1, this->buffer1_size, this->stream));
    this->buffer2_size = BUFFER_INIT_SIZE;
    CHECK_CUDA(hipMallocAsync((void**) &this->buffer2, this->buffer2_size, this->stream));

    // Create MatDesc
    CHECK_CUSPARSE( hipsparseCreateMatDescr(&this->matDescr) ); 
    hipsparseSetMatType(this->matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(this->matDescr, HIPSPARSE_INDEX_BASE_ZERO);

    // Set Spgemm
    if (always_use_safe)
        this->spgemm = &CuSparseCore::_spgemm_safe;
    else
        this->spgemm = &CuSparseCore::_spgemm;

#ifdef TIME_MEASUREMENT
    this->resetTimer();
#endif // TIME_MEASUREMENT
}

CuSparseCore::~CuSparseCore()
{
    // Free buffer
    CHECK_CUDA(hipFreeAsync(this->buffer1, this->stream));
    CHECK_CUDA(hipFreeAsync(this->buffer2, this->stream));

    // Destroy handle
    CHECK_CUSPARSE(hipsparseDestroy(this->handle));
    CHECK_CUSPARSE(hipsparseDestroy(this->associate_handle));

    // Destroy stream
    CHECK_CUDA(hipStreamDestroy(this->stream));
    CHECK_CUDA(hipStreamDestroy(this->associate_stream));

    // Destory MatDesc
    CHECK_CUSPARSE( hipsparseDestroyMatDescr(this->matDescr) );
}

#ifdef TIME_MEASUREMENT
void CuSparseCore::resetTimer()
{
    this->total_nnz = 0;
    this->spgemm_count = 0;
    this->spgemm_safe_count = 0;
    this->spgemm_timer = std::chrono::duration<double>::zero();
    this->spgemm_safe_timer = std::chrono::duration<double>::zero();
}

void CuSparseCore::printTimer()
{
    // Print counter
    if (this->total_nnz > 0)
        std::cout << "Total NNZ:\t" << this->total_nnz  << std::endl;
    if (this->spgemm_count > 0)
        std::cout << "SPGEMM_DEFAULT:\t" << this->spgemm_count << "\tTime: " << this->spgemm_timer.count() << std::endl;
    if (this->spgemm_safe_count > 0)
        std::cout << "SPGEMM_SAFE:\t" << this->spgemm_safe_count << "\tTime: " << this->spgemm_safe_timer.count() << std::endl;
}
#endif // TIME_MEASUREMENT


cuSparseMatrix *CuSparseCore::_spgemm(
    cuSparseMatrix &matA, 
    cuSparseMatrix &matB, 
    value_type_t alpha /*=1.0f*/)
{
#ifdef DEBUG
    assert(matA.d_csrOffsets != NULL);
    assert(matA.d_columns != NULL);
    assert(matA.d_values != NULL);
    assert(matB.d_csrOffsets != NULL);
    assert(matB.d_columns != NULL);
    assert(matB.d_values != NULL);
#endif
    // Check matDescr
    if ((matA.matDescr == 0))
    {
        CHECK_CUSPARSE(hipsparseCreateCsr(&matA.matDescr, 
            matA.num_rows, matA.num_cols, matA.nnz,
            matA.d_csrOffsets, matA.d_columns, matA.d_values,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }
    if ((matB.matDescr == 0))
    {
        CHECK_CUSPARSE(hipsparseCreateCsr(&matB.matDescr, 
            matB.num_rows, matB.num_cols, matB.nnz,
            matB.d_csrOffsets, matB.d_columns, matB.d_values,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }

    // Get handle
    hipsparseHandle_t &handle = this->handle;
    hipStream_t &stream = this->stream;

    // We are perfroming the simpliest computation: matC = matA*matB
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    value_type_t beta = float2value(0.0f);

    // Prepare tmp buffers
    size_t bufferSize1 = 0, bufferSize2 = 0;

#ifdef TIME_MEASUREMENT
    // Compute the intermediate product of A * B
    auto begin_time = std::chrono::high_resolution_clock::now();
#endif // TIME_MEASUREMENT

    // Init cusparse SpGEMM descriptor
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc));

    // Init the result matrix
    cuSparseMatrix *matResultPtr = new cuSparseMatrix(matA.num_rows, matB.num_cols);

    // Create Desc
    CHECK_CUSPARSE(hipsparseCreateCsr(&matResultPtr->matDescr, 
        matResultPtr->num_rows, matResultPtr->num_cols, 0,
        NULL, NULL, NULL,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // Ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA.matDescr, matB.matDescr, &beta, matResultPtr->matDescr,
                                      COMPUTE_TYPE, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) );

    if (UNLIKELY(this->buffer1_size < bufferSize1))
    {
        #ifdef DEBUG
            printf("spgemm: Buffer 1 is not large enough: current size is %lu but %lu is needed.\n", this->buffer1_size, bufferSize1);
        #endif
        CHECK_CUDA(hipFreeAsync(this->buffer1, stream));
        CHECK_CUDA(hipMallocAsync((void**) &this->buffer1, bufferSize1, stream));
        this->buffer1_size = bufferSize1;
    }

    // Inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA.matDescr, matB.matDescr, &beta, matResultPtr->matDescr,
                                      COMPUTE_TYPE, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, this->buffer1) );
                                
    // Ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA.matDescr, matB.matDescr, &beta, matResultPtr->matDescr,
                               COMPUTE_TYPE, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL));

    if (UNLIKELY(this->buffer2_size < bufferSize2))
    {
        #ifdef DEBUG
            printf("spgemm: Buffer 2 is not large enough: current size is %lu but %lu is needed.\n", this->buffer2_size, bufferSize2);
        #endif
        CHECK_CUDA(hipFreeAsync(this->buffer2, stream));
        CHECK_CUDA(hipMallocAsync((void**) &this->buffer2, bufferSize2, stream));
        this->buffer2_size = bufferSize2;
    }

    auto errorId = hipsparseSpGEMM_compute(handle, opA, opB,
            &alpha, matA.matDescr, matB.matDescr, &beta, matResultPtr->matDescr,
            COMPUTE_TYPE, HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc, &bufferSize2, this->buffer2);

    // cusparseSpGEMM() has a limitation which uses too much memory.
    // If it happens, we fallback to a deprecated api, hipsparseScsrgemm2().
    // See https://github.com/NVIDIA/CUDALibrarySamples/issues/38 for details
    if (UNLIKELY(errorId != HIPSPARSE_STATUS_SUCCESS))
    {
        if (LIKELY(errorId == HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES))
        {
            #ifdef DEBUG
                printf("spgemm: Insufficient resources! Fallback to spgemm_safe()!\n");
            #endif
            // Destory SpGEMM descriptor
            CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) );
            
            // Clean-up
            delete matResultPtr;
            
            // Switch routine
            matResultPtr = this->_spgemm_safe(matA, matB, alpha);
            
            // Set Sparse Desc for _spgemm
            CHECK_CUSPARSE(hipsparseCreateCsr(&matResultPtr->matDescr, 
                matResultPtr->num_rows, matResultPtr->num_cols, matResultPtr->nnz,
                matResultPtr->d_csrOffsets, matResultPtr->d_columns, matResultPtr->d_values,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

            // Return
            return matResultPtr;
        }
        else
        {
            // Not because of insufficient res. Raise this error.
            CHECK_CUSPARSE(errorId);
        }
    }

    // Get result matrix non-zero entries
    matResultPtr->nnz = 0;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matResultPtr->matDescr, &matResultPtr->num_rows, 
        &matResultPtr->num_cols, &matResultPtr->nnz) );

    // Allocate result matrix
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_csrOffsets, (matA.num_rows + 1) * sizeof(int),stream) );
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_columns, matResultPtr->nnz * sizeof(int),     stream) );
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_values,  matResultPtr->nnz * sizeof(float),   stream) );

    // Update result matrix with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matResultPtr->matDescr, matResultPtr->d_csrOffsets, matResultPtr->d_columns, matResultPtr->d_values) );
    
    // Copy the final products to the result matrix
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA.matDescr, matB.matDescr, &beta, matResultPtr->matDescr,
                            COMPUTE_TYPE, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) );

    // Destory descriptor
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) );

#ifdef TIME_MEASUREMENT
    auto end_time = std::chrono::high_resolution_clock::now();
#endif // TIME_MEASUREMENT

#ifdef TIME_MEASUREMENT
    // Update counter
    this->total_nnz += matResultPtr->nnz;
    this->spgemm_count++;
    this->spgemm_timer += end_time - begin_time;
#endif 

    // Return result
    return matResultPtr;
}

cuSparseMatrix *CuSparseCore::_spgemm_safe(
    cuSparseMatrix &matA, 
    cuSparseMatrix &matB, 
    value_type_t alpha /*=1.0f*/)
{
    // Get handle
    hipsparseHandle_t &handle = this->handle;
    hipStream_t &stream = this->stream;

    // Init result
    cuSparseMatrix *matResultPtr = new cuSparseMatrix(matA.num_rows, matB.num_cols);

#ifdef TIME_MEASUREMENT
    auto begin_time = std::chrono::high_resolution_clock::now();
#else
    assert(matA.num_cols == matB.num_rows);
#endif

    // step 1: create an opaque structure
    csrgemm2Info_t info = NULL;
    CHECK_CUSPARSE(hipsparseCreateCsrgemm2Info(&info));

    // step 2: allocate buffer for csrgemm2Nnz and csrgemm2
    size_t bufferSize1;
    CHECK_CUSPARSE(hipsparseScsrgemm2_bufferSizeExt(
        handle, matA.num_rows, matB.num_cols, matA.num_cols, &alpha,
        this->matDescr, matA.nnz, matA.d_csrOffsets, matA.d_columns,
        this->matDescr, matB.nnz, matB.d_csrOffsets, matB.d_columns,
        NULL,  /*beta*/
        this->matDescr, 0, NULL, NULL, //nnzD, csrRowPtrD, csrColIndD,
        info,
        &bufferSize1));
    
    if (UNLIKELY(this->buffer1_size < bufferSize1))
    {
        printf("spgemm_safe: Buffer 1 is not large enough: current size is %lu but %lu is needed.\n", this->buffer1_size, bufferSize1);
        CHECK_CUDA(hipFreeAsync(this->buffer1, stream));
        CHECK_CUDA(hipMallocAsync((void**) &this->buffer1, bufferSize1, stream));
        this->buffer1_size = bufferSize1;
    }
    void *&buffer = this->buffer1;

    // step 3: compute csrRowPtr
    int nnzTotalDevHostPtr = -1;
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_csrOffsets, (matA.num_rows + 1) * sizeof(int),stream) );  
    
    CHECK_CUSPARSE(hipsparseXcsrgemm2Nnz(
            handle, matA.num_rows, matB.num_cols, matA.num_cols,
            this->matDescr, matA.nnz, matA.d_csrOffsets, matA.d_columns,
            this->matDescr, matB.nnz, matB.d_csrOffsets, matB.d_columns,
            this->matDescr, 0, NULL, NULL,//nnzD, csrRowPtrD, csrColIndD,
            this->matDescr, matResultPtr->d_csrOffsets, &nnzTotalDevHostPtr,
            info, buffer));
    matResultPtr->nnz = (int64_t) nnzTotalDevHostPtr;

    // step 4: finish sparsity pattern and value
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_columns, matResultPtr->nnz * sizeof(int),     stream));
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_values,  matResultPtr->nnz * sizeof(float),   stream));

    CHECK_CUSPARSE(hipsparseScsrgemm2(
        handle, matA.num_rows, matB.num_cols, matA.num_cols, &alpha,
        this->matDescr, matA.nnz, matA.d_values, matA.d_csrOffsets, matA.d_columns,
        this->matDescr, matB.nnz, matB.d_values, matB.d_csrOffsets, matB.d_columns,
        NULL, // beta
        this->matDescr, 0, NULL, NULL, NULL, //nnzD, csrValD, csrRowPtrD, csrColIndD,
        this->matDescr, matResultPtr->d_values, matResultPtr->d_csrOffsets, matResultPtr->d_columns,
        info, buffer));

    // Destory Csrgemm Info
    CHECK_CUSPARSE(hipsparseDestroyCsrgemm2Info(info));

#ifdef TIME_MEASUREMENT
    auto end_time = std::chrono::high_resolution_clock::now();
    this->total_nnz += matResultPtr->nnz;
    this->spgemm_safe_count++;
    this->spgemm_safe_timer += end_time - begin_time;
#endif

    return matResultPtr;
}

cuSparseMatrix* CuSparseCore::add(
    cuSparseMatrix &matA, 
    cuSparseMatrix &matB, 
    float alpha/* = 1.0f */, 
    float belta /* = 1.0f */,
    bool use_associate /*= false*/)
{
#ifdef DEBUG
    assert(matA.d_csrOffsets != NULL);
    assert(matA.d_columns != NULL);
    assert(matA.d_values != NULL);
    assert(matB.d_csrOffsets != NULL);
    assert(matB.d_columns != NULL);
    assert(matB.d_values != NULL);
#endif
    // Get handle
    hipsparseHandle_t &handle = use_associate ? this->associate_handle : this->handle;
    hipStream_t &stream = use_associate ? this->associate_stream : this->stream;
    void *&buffer = use_associate ? this->buffer2 : this->buffer1;
    size_t &buffer_size = use_associate ? this->buffer2_size : this->buffer1_size;

    // Init the result matrix
    cuSparseMatrix *matResultPtr = new cuSparseMatrix(matA.num_rows, matA.num_cols);

    // Malloc row ptr for the result matrix
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_csrOffsets, (matA.num_rows + 1) * sizeof(int), stream) );
    
    // Get buffer size
    size_t requiredBufferSize;
    CHECK_CUSPARSE(
        hipsparseScsrgeam2_bufferSizeExt(handle, matResultPtr->num_rows, matResultPtr->num_cols, 
            &alpha, this->matDescr, matA.nnz, matA.d_values, matA.d_csrOffsets, matA.d_columns,
            &belta, this->matDescr, matB.nnz, matB.d_values, matB.d_csrOffsets, matB.d_columns,
            this->matDescr, matResultPtr->d_values, matResultPtr->d_csrOffsets, matResultPtr->d_columns, 
            &requiredBufferSize    
    ));

    if (UNLIKELY(buffer_size < requiredBufferSize))
    {
        #ifdef DEBUG
            printf("add: Buffer 1 is not large enough: current size is %lu but %lu is needed.\n", buffer_size, requiredBufferSize);
        #endif
        CHECK_CUDA(hipFreeAsync(buffer, stream));
        CHECK_CUDA(hipMallocAsync((void**) &buffer, requiredBufferSize, stream));
        buffer_size = requiredBufferSize;
    }
    int nnz_result = -1;

    // if (!
    CHECK_CUSPARSE(
        hipsparseXcsrgeam2Nnz(handle, matResultPtr->num_rows, matResultPtr->num_cols, 
            this->matDescr, matA.nnz, matA.d_csrOffsets, matA.d_columns,
            this->matDescr, matB.nnz, matB.d_csrOffsets, matB.d_columns,
            this->matDescr, matResultPtr->d_csrOffsets, &nnz_result,
            buffer));
    // )
    // {
    //     matResultPtr->nnz = 0;
    //     CHECK_CUDA(hipFreeAsync(matResultPtr->d_csrOffsets, stream));
    //     matResultPtr->d_csrOffsets = NULL;
    //     return matResultPtr;
    // };
    matResultPtr->nnz = (int64_t) nnz_result;
    
    // Malloc column and values
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_columns, matResultPtr->nnz * sizeof(int),     stream));
    CHECK_CUDA( hipMallocAsync((void**) &matResultPtr->d_values,  matResultPtr->nnz * sizeof(float),   stream));
    
    // Finish computation
    CHECK_CUSPARSE(
        hipsparseScsrgeam2(handle, matResultPtr->num_rows, matResultPtr->num_cols, 
            &alpha, this->matDescr, matA.nnz, matA.d_values, matA.d_csrOffsets, matA.d_columns,
            &belta, this->matDescr, matB.nnz, matB.d_values, matB.d_csrOffsets, matB.d_columns,
            this->matDescr, matResultPtr->d_values, matResultPtr->d_csrOffsets, matResultPtr->d_columns, 
            buffer 
    ));

    return matResultPtr;
}

void CuSparseCore::sync()
{
    hipStreamSynchronize(this->stream);
    hipStreamSynchronize(this->associate_stream);
}


// void CuSparseCore::compress(int streamId, cuSparseMatrix &matA)
// {
//     float tol = 0.5;

//     // Get handle
//     hipsparseHandle_t handle = this->streamPool->getCusparseHandle(streamId);
//     hipStream_t stream = this->streamPool->getStreamFromPool(streamId);
    
//     // Malloc a temp buffer to store nnz per row
//     int *nnz_per_row;
//     int *new_nnz;
//     CHECK_CUDA( hipMallocManaged((void**) &nnz_per_row, (matA.num_rows+1) * sizeof(int)) );
//     CHECK_CUDA( hipMallocManaged((void**) &new_nnz, sizeof(int)) );

//     memset( nnz_per_row, 0, (matA.num_rows+1) * sizeof(int));

//     // Get nnz per row
//     CHECK_CUSPARSE( hipsparseSnnz_compress(
//         handle, matA.num_rows, this->matDescr, 
//         matA.d_values, matA.d_csrOffsets, nnz_per_row,
//         new_nnz, tol));
    
//     // Malloc new space for the compressed matrix
//     float *csrVal;
//     int *csrRowPtr;
//     int *csrColInd;
//     CHECK_CUDA( hipMallocAsync( &csrVal, sizeof(float) * (*new_nnz), stream));
//     CHECK_CUDA( hipMallocAsync( &csrRowPtr, sizeof(int) * (matA.num_rows+1), stream));
//     CHECK_CUDA( hipMallocAsync( &csrColInd, sizeof(int) * (*new_nnz), stream));

//     CHECK_CUSPARSE( hipsparseScsr2csr_compress( 
//         handle,matA.num_rows, matA.num_cols, this->matDescr, 
//         matA.d_values, matA.d_columns, matA.d_csrOffsets,
//         matA.nnz,  nnz_per_row,
//         csrVal, csrRowPtr, csrColInd, tol));
    
//     CHECK_CUDA( hipMemcpyAsync(&matA.nnz, new_nnz, sizeof(int),
//         hipMemcpyDeviceToHost, stream) );

//     // Free malloced memory
//     CHECK_CUDA( hipFreeAsync(matA.d_values, stream) );
//     CHECK_CUDA( hipFreeAsync(matA.d_columns, stream) );
//     CHECK_CUDA( hipFreeAsync(matA.d_csrOffsets, stream) );

//     // Set pointers
//     matA.d_values = csrVal;
//     matA.d_columns = csrColInd;
//     matA.d_csrOffsets = csrRowPtr;
// }